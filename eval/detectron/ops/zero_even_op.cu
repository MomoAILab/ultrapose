#include "hip/hip_runtime.h"
/**
# Copyright (c) Facebook, Inc. and its affiliates.
# All rights reserved.
#
# This source code is licensed under the license found in the
# LICENSE file in the root directory of this source tree.
 */

#include "caffe2/core/context_gpu.h"

#include "zero_even_op.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void SetEvenIndsToVal(size_t num_even_inds, T val, T* data) {
  CUDA_1D_KERNEL_LOOP(i, num_even_inds) {
    data[i << 1] = val;
  }
}

} // namespace

template <>
bool ZeroEvenOp<float, HIPContext>::RunOnDevice() {
  // Retrieve the input tensor.
  const auto& X = Input(0);
  CAFFE_ENFORCE(X.ndim() == 1);

  // Initialize the output tensor to a copy of the input tensor.
  auto* Y = Output(0);
  Y->CopyFrom(X);

  // Set output elements at even indices to zero.
  auto output_size = Y->size();

  if (output_size > 0) {
    size_t num_even_inds = output_size / 2 + output_size % 2;
    SetEvenIndsToVal<float>
        <<<CAFFE_GET_BLOCKS(num_even_inds),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            num_even_inds,
            0.0f,
            Y->mutable_data<float>());
  }

  return true;
}

REGISTER_HIP_OPERATOR(ZeroEven, ZeroEvenOp<float, HIPContext>);

} // namespace caffe2
